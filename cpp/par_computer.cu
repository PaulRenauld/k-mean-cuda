#include "hip/hip_runtime.h"
#include "par_computer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <iostream>
#include <set>
#include <iterator>
#include <random>

// #define DEBUG

#ifdef DEBUG
#define cudaCheckKernelError() cudaCheckError( hipDeviceSynchronize () )
#define cudaCheckError(ans)  cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char
*
file, int line, bool abort=true)
{
if (code != hipSuccess)
{
fprintf(stderr, "CUDA Error: %s at %s:%d\n",
hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#else
#define cudaCheckKernelError() 
#define cudaCheckError(ans) ans
#endif



////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {
  size_t k;
  size_t n;
  Dataset dataset;
  ClusterPosition clusters;
  ClusterAccumulator *accumulators;
  unsigned short *cluster_for_point;
};

__constant__ GlobalConstants cuConstParams;

__device__ float distance_square(Point first, Point second) {
  float diff_x = first.x - second.x;
  float diff_y = first.y - second.y;
  return diff_x * diff_x + diff_y * diff_y;
}

__global__ void kernel_update_cluster(bool* change) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= cuConstParams.n) return;

  Point datapoint = cuConstParams.dataset[index];
  Point f = cuConstParams.clusters[0];
  // float minimum = datapoint.distance_squared_to(f);
  float minimum = distance_square(datapoint, f);
  unsigned short index_min = 0;

  for (unsigned short j = 1; j < cuConstParams.k; j++) {
    float distance = distance_square(datapoint, cuConstParams.clusters[j]);
    if (distance < minimum) {
      minimum = distance;
      index_min = j;
    }
  }

  if (cuConstParams.cluster_for_point[index] != index_min) {
    // printf("New cluster for point: %u \n", index_min);
    cuConstParams.cluster_for_point[index] = index_min;
    *change = true;
  }
  return;
}

__global__ void kernel_update_cluster_accumulators() {
  extern __shared__ ClusterAccumulator accs[];
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  for (int k = threadIdx.x; k < cuConstParams.k; k += blockDim.x) {
    accs[k].x = 0;
    accs[k].y = 0;
    accs[k].count = 0;
  }
  __syncthreads();

  if (index < cuConstParams.n) {
    Point point = cuConstParams.dataset[index];
    size_t cluster = cuConstParams.cluster_for_point[index];
    atomicAdd(&accs[cluster].x, point.x);
    atomicAdd(&accs[cluster].y, point.y);
    atomicAdd(&accs[cluster].count, 1);
  }

  __syncthreads();
  for (int k = threadIdx.x; k < cuConstParams.k; k += blockDim.x) {
    atomicAdd(&cuConstParams.accumulators[k].x, accs[k].x);
    atomicAdd(&cuConstParams.accumulators[k].y, accs[k].y);
    atomicAdd(&cuConstParams.accumulators[k].count, accs[k].count);
  }
}

__global__ void kernel_update_cluster_positions() {
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  if (k >= cuConstParams.k) return;
  ClusterAccumulator acc = cuConstParams.accumulators[k];
  cuConstParams.clusters[k].x = acc.x / acc.count;
  cuConstParams.clusters[k].y = acc.y / acc.count;
  cuConstParams.accumulators[k].x = 0;
  cuConstParams.accumulators[k].y = 0;
  cuConstParams.accumulators[k].count = 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////

par_computer::par_computer(size_t k, size_t n, Dataset dataset) : kmean_computer(k, n, dataset) {
  cudaCheckError( hipMalloc(&cudaDeviceDataset, sizeof(Point) * n) );
  cudaCheckError( hipMemcpy(cudaDeviceDataset, dataset, sizeof(Point) * n, hipMemcpyHostToDevice) );

  cudaCheckError( hipMalloc(&cudaDeviceClusters, sizeof(Point) * k) );

  cudaCheckError( hipMalloc(&cuda_device_cluster_for_point, sizeof(unsigned short) * n) );

  cudaCheckError( hipMalloc(&clusterAccumulators, sizeof(ClusterAccumulator) * k) );

  GlobalConstants params;
  params.k = k;
  params.n = n;
  params.dataset = cudaDeviceDataset;
  params.clusters = cudaDeviceClusters;
  params.cluster_for_point = cuda_device_cluster_for_point;
  params.accumulators = clusterAccumulators;

  cudaCheckError( hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants)) );

  cudaCheckError( hipDeviceSynchronize() );
}

par_computer::~par_computer() {
  hipFree(cudaDeviceDataset);
  hipFree(cudaDeviceClusters);
  hipFree(cuda_device_cluster_for_point);
  hipFree(clusterAccumulators);
}

void par_computer::init_starting_clusters() {
  std::default_random_engine generator;
  std::uniform_int_distribution<size_t> distribution(0, n - 1);

  std::set<size_t, std::greater<size_t>> positions;
  while (positions.size() < k) {
    positions.insert(distribution(generator));
  }

  size_t i = 0;
  for (auto index: positions) {
    clusters[i++] = dataset[index];
  }

  // Clear the accumulators
  dim3 blockDim(256, 1);
  dim3 gridDim((k + blockDim.x - 1) / blockDim.x);
  kernel_update_cluster_positions<<<gridDim, blockDim>>>();

  cudaCheckError( hipMemcpy(cudaDeviceClusters, clusters, sizeof(Point) * k, hipMemcpyHostToDevice) );
}

void par_computer::update_cluster_positions() {
  dim3 blockDim(256, 1);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x);
  kernel_update_cluster_accumulators<<<gridDim, blockDim, k * sizeof(ClusterAccumulator)>>>();
  cudaCheckKernelError();

  gridDim = dim3((k + blockDim.x - 1) / blockDim.x);
  kernel_update_cluster_positions<<<gridDim, blockDim>>>();
  cudaCheckKernelError();
}

bool par_computer::update_cluster_for_point() {
  dim3 blockDim(256, 1);
  dim3 gridDim((n + blockDim.x - 1) / blockDim.x, 1);

  bool* change;
  cudaCheckError( hipMalloc(&change, sizeof(bool)) );

  kernel_update_cluster<<<gridDim, blockDim>>>(change);
  cudaCheckKernelError();

  bool changeHost = false;
  cudaCheckError( hipMemcpy(&changeHost, change, sizeof(bool), hipMemcpyDeviceToHost) );

  return changeHost;
}


void par_computer::after_converge() {
  cudaCheckError( hipMemcpy(clusters, cudaDeviceClusters, sizeof(Point) * k, hipMemcpyDeviceToHost) );
  cudaCheckError( hipMemcpy(cluster_for_point, cuda_device_cluster_for_point, sizeof(unsigned short) * n, hipMemcpyDeviceToHost) );
}
