#include "par_computer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <set>
#include <iterator>
#include <random>

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////
struct GlobalConstants {
  size_t k;
  size_t n;
  Dataset dataset;
  ClusterPosition clusters;
  unsigned short *cluster_for_point;
};

__constant__ GlobalConstants cuConstParams;

///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////

void par_computer::par_computer(size_t k, size_t n, Dataset dataset) k(k), n(n), dataset(dataset){
  clusters = new Point[k];
  cluster_for_point = new unsigned short[n];

  init_starting_clusters(); 

  hipMalloc(&cudaDeviceDataset, sizeof(Point) * n);
  hipMemcpy(cudaDeviceDataset, dataset, sizeof(Point) * n, hipMemcpyHostToDevice);

  hipMalloc(&cudaDeviceClusters, sizeof(Point) * k);
  hipMemcpy(cudaDeviceClusters, clusters, sizeof(Point) * k, hipMemcpyHostToDevice);

  hipMalloc(&cuda_device_cluster_for_point, sizeof(unsigned short) * n);

  GlobalConstants params;
  params.k = k;
  params.n = n;
  params.dataset = cudaDeviceDataset;
  params.clusters = cudaDeviceClusters;
  params.cluster_for_point = cuda_device_cluster_for_point;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

}

void par_computer::~par_computer() {
  delete[] clusters;
  delete[] cluster_for_point;

  hipFree(cudaDeviceDataset);
  hipFree(cudaDeviceClusters);
  hipFree(cuda_device_cluster_for_point);
}

void par_computer::init_starting_clusters() {
  std::default_random_engine generator;
  std::uniform_int_distribution<size_t> distribution(0, n - 1);
  auto random_index = std::bind ( distribution, generator );

  std::set<size_t, std::greater<>> positions;
  while (positions.size() < k) {
    positions.insert(random_index());
  }

  size_t i = 0;
  for (auto index: positions) {
    clusters[i++] = dataset[index];
  }
}

void par_computer::update_cluster_positions() {

}

bool par_computer::update_cluster_for_point() {

}

ClusterPosition par_computer::converge() {
  while (update_cluster_for_point()) {
    update_cluster_positions();
  }

  hipMemcpy(clusters, cudaDeviceClusters, sizeof(Point) * k, hipMemcpyDeviceToHost);

  return clusters;
}