#include "par_computer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <set>
#include <iterator>
#include <random>

////////////////////////////////////////////////////////////////////////////////////////
// All cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////
struct GlobalConstants {
  size_t k;
  size_t n;
  Dataset dataset;
  ClusterPosition clusters;
  unsigned short *cluster_for_point;
};

__constant__ GlobalConstants cuConstParams;

///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////

par_computer::par_computer(size_t k, size_t n, Dataset dataset) : kmean_computer(k, n, dataset) {
  clusters = new Point[k];
  cluster_for_point = new unsigned short[n];

  init_starting_clusters(); 

  hipMalloc(&cudaDeviceDataset, sizeof(Point) * n);
  hipMemcpy(cudaDeviceDataset, dataset, sizeof(Point) * n, hipMemcpyHostToDevice);

  hipMalloc(&cudaDeviceClusters, sizeof(Point) * k);
  hipMemcpy(cudaDeviceClusters, clusters, sizeof(Point) * k, hipMemcpyHostToDevice);

  hipMalloc(&cuda_device_cluster_for_point, sizeof(unsigned short) * n);

  GlobalConstants params;
  params.k = k;
  params.n = n;
  params.dataset = cudaDeviceDataset;
  params.clusters = cudaDeviceClusters;
  params.cluster_for_point = cuda_device_cluster_for_point;

  hipMemcpyToSymbol(HIP_SYMBOL(cuConstParams), &params, sizeof(GlobalConstants));

}

par_computer::~par_computer() {
  delete[] clusters;
  delete[] cluster_for_point;

  hipFree(cudaDeviceDataset);
  hipFree(cudaDeviceClusters);
  hipFree(cuda_device_cluster_for_point);
}

void par_computer::init_starting_clusters() {
  std::default_random_engine generator;
  std::uniform_int_distribution<size_t> distribution(0, n - 1);

  std::set<size_t, std::greater<size_t>> positions;
  while (positions.size() < k) {
    positions.insert(distribution(generator));
  }

  size_t i = 0;
  for (auto index: positions) {
    clusters[i++] = dataset[index];
  }
}

void par_computer::update_cluster_positions() {

}

bool par_computer::update_cluster_for_point() {

}

ClusterPosition par_computer::converge() {
  while (update_cluster_for_point()) {
    update_cluster_positions();
  }

  hipMemcpy(clusters, cudaDeviceClusters, sizeof(Point) * k, hipMemcpyDeviceToHost);

  return clusters;
}